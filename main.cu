#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <GL/glew.h>
#include <GL/freeglut.h>
#include "config.h"
#include "util.cuh"
#include "draw.cuh"
#include "math.cuh"

using namespace std;

// Capitalised because they are effectively constant
int CPU_CELL_NEIGHBOURHOOD_COMBINATIONS = -1;
__constant__ int GPU_CELL_NEIGHBOURHOOD_COMBINATIONS = -1;
int CPU_RULESET_SIZE = -1;
__constant__ int GPU_RULESET_SIZE = -1;

__inline__ __host__ __device__ int get_cell_neighbourhood_combinations() {
#ifdef __CUDA_ARCH__
    return GPU_CELL_NEIGHBOURHOOD_COMBINATIONS;
#else
    return CPU_CELL_NEIGHBOURHOOD_COMBINATIONS;
#endif
}

__inline__ __host__ __device__ int get_ruleset_size() {
#ifdef __CUDA_ARCH__
    return GPU_RULESET_SIZE;
#else
    return CPU_RULESET_SIZE;
#endif
}

__inline__ __host__ __device__ void get_neighbours(i32 x, i32 y, i32vec2 neighbours[CELL_NEIGHBOURHOOD_SIZE]) {
#if GRID_GEOMETRY == GRID_GEOMETRY_SQUARE
    #if CELL_NEIGHBOURHOOD_TYPE == CELL_NEIGHBOURHOOD_TYPE_VERTEX
    neighbours[0] = make_i32vec2(-1, -1);
    neighbours[1] = make_i32vec2( 0, -1);
    neighbours[2] = make_i32vec2( 1, -1);
    neighbours[3] = make_i32vec2(-1,  0);
    neighbours[4] = make_i32vec2( 1,  0);
    neighbours[5] = make_i32vec2(-1,  1);
    neighbours[6] = make_i32vec2( 0,  1);
    neighbours[7] = make_i32vec2( 1,  1);
    #elif CELL_NEIGHBOURHOOD_TYPE == CELL_NEIGHBOURHOOD_TYPE_EDGE
    neighbours[0] = make_i32vec2(-1,  0);
    neighbours[1] = make_i32vec2( 0, -1);
    neighbours[2] = make_i32vec2( 1,  0);
    neighbours[3] = make_i32vec2( 0,  1);
    #endif
#elif GRID_GEOMETRY == GRID_GEOMETRY_TRIANGLE
    bool pointing_up = (x + y) % 2 == 0;
    #if CELL_NEIGHBOURHOOD_TYPE == CELL_NEIGHBOURHOOD_TYPE_VERTEX
    if (pointing_up) {
        neighbours[ 0] = make_i32vec2(-1,  0);
        neighbours[ 1] = make_i32vec2(-1, -1);
        neighbours[ 2] = make_i32vec2( 0, -1);
        neighbours[ 3] = make_i32vec2( 1, -1);
        neighbours[ 4] = make_i32vec2( 1,  0);
        neighbours[ 5] = make_i32vec2( 2,  0);
        neighbours[ 6] = make_i32vec2( 2,  1);
        neighbours[ 7] = make_i32vec2( 1,  1);
        neighbours[ 8] = make_i32vec2( 0,  1);
        neighbours[ 9] = make_i32vec2(-1,  1);
        neighbours[10] = make_i32vec2(-2,  1);
        neighbours[11] = make_i32vec2(-2,  0);
    } else {
        neighbours[ 0] = make_i32vec2(-1,  0);
        neighbours[ 1] = make_i32vec2(-2,  0);
        neighbours[ 2] = make_i32vec2(-2, -1);
        neighbours[ 3] = make_i32vec2(-1, -1);
        neighbours[ 4] = make_i32vec2( 0, -1);
        neighbours[ 5] = make_i32vec2( 1, -1);
        neighbours[ 6] = make_i32vec2( 2, -1);
        neighbours[ 7] = make_i32vec2( 2,  0);
        neighbours[ 8] = make_i32vec2( 1,  0);
        neighbours[ 9] = make_i32vec2( 1,  1);
        neighbours[10] = make_i32vec2( 0,  1);
        neighbours[11] = make_i32vec2(-1,  1);
    }
    #elif CELL_NEIGHBOURHOOD_TYPE == CELL_NEIGHBOURHOOD_TYPE_EDGE
    if (pointing_up) {
        neighbours[0] = make_i32vec2(-1,  0);
        neighbours[1] = make_i32vec2( 1,  0);
        neighbours[2] = make_i32vec2( 0,  1);
    } else {
        neighbours[0] = make_i32vec2(-1,  0);
        neighbours[1] = make_i32vec2( 1,  0);
        neighbours[2] = make_i32vec2( 0, -1);
    }
    #endif
#elif GRID_GEOMETRY == GRID_GEOMETRY_HEXAGON
    bool row_even = y % 2 == 0;

    if (row_even) {
        neighbours[0] = make_i32vec2(-1,  0);
        neighbours[1] = make_i32vec2(-1, -1);
        neighbours[2] = make_i32vec2( 0, -1);
        neighbours[3] = make_i32vec2( 1,  0);
        neighbours[4] = make_i32vec2( 0,  1);
        neighbours[5] = make_i32vec2(-1,  1);
    } else {
        neighbours[0] = make_i32vec2(-1,  0);
        neighbours[1] = make_i32vec2( 0, -1);
        neighbours[2] = make_i32vec2( 1, -1);
        neighbours[3] = make_i32vec2( 1,  0);
        neighbours[4] = make_i32vec2( 1,  1);
        neighbours[5] = make_i32vec2( 0,  1);
    }
#endif
}

void print_configuration() {
    printf("\nConfiguration:\n");
    printf("\tGrid width: %d\n", GRID_WIDTH);
    printf("\tGrid height: %d\n", GRID_HEIGHT);

    if (GRID_GEOMETRY == GRID_GEOMETRY_SQUARE) {
        printf("\tGrid geometry: Square\n");
    } else if (GRID_GEOMETRY == GRID_GEOMETRY_TRIANGLE) {
        printf("\tGrid geometry: Triangle\n");
    } else if (GRID_GEOMETRY == GRID_GEOMETRY_HEXAGON) {
        printf("\tGrid geometry: Hexagon\n");
    } else {
        printf("\tGrid geometry: Invalid, aborting...\n");
        exit(1);
    }

    if (CELL_NEIGHBOURHOOD_TYPE == CELL_NEIGHBOURHOOD_TYPE_VERTEX) {
        printf("\tCell neighbourhood type: Vertex\n");
    } else if (CELL_NEIGHBOURHOOD_TYPE == CELL_NEIGHBOURHOOD_TYPE_EDGE) {
        printf("\tCell neighbourhood type: Edge\n");
    } else {
        printf("\tCell neighbourhood type: Invalid, aborting...\n");
        exit(1);
    }

    printf("\tCell neighbourhood size: %d\n", CELL_NEIGHBOURHOOD_SIZE);
    printf("\tCell states: %d\n", CELL_STATES);

    CPU_CELL_NEIGHBOURHOOD_COMBINATIONS = compute_neighbouring_state_combinations(CELL_NEIGHBOURHOOD_SIZE, CELL_STATES);
    hipMemcpyToSymbol(HIP_SYMBOL(GPU_CELL_NEIGHBOURHOOD_COMBINATIONS), &CPU_CELL_NEIGHBOURHOOD_COMBINATIONS, sizeof(int));
    printf("\tCell neighbourhood combinations: %d (with a combinatorial number system, %ld with simple indexing)\n", CPU_CELL_NEIGHBOURHOOD_COMBINATIONS, powli(CELL_NEIGHBOURHOOD_SIZE, CELL_STATES));

    CPU_RULESET_SIZE = compute_ruleset_size(CELL_NEIGHBOURHOOD_SIZE, CELL_STATES);
    hipMemcpyToSymbol(HIP_SYMBOL(GPU_RULESET_SIZE), &CPU_RULESET_SIZE, sizeof(int));
    printf("\tRuleset size: %d (with a combinatorial number system, %ld with simple indexing)\n", CPU_RULESET_SIZE, CELL_STATES * powli(CELL_NEIGHBOURHOOD_SIZE, CELL_STATES));
    printf("\tshared_subgrid_margin: %d\n", SHARED_SUBGRID_MARGIN);
    printf("\tshared_subgrid_length: %d\n", SHARED_SUBGRID_LENGTH);
    printf("\tshared_subgrid_area: %d\n", SHARED_SUBGRID_AREA);
    printf("\tshared_subgrid_load_iterations: %d\n", SHARED_SUBGRID_LOAD_ITERATIONS);
    printf("\n");

/*     if (GRID_AREA >= (1l << 16)) { */
/*         fprintf(stderr, "Grid size (%d) exceeds max value (%d).\n", GRID_AREA, (1l << 16)); */
/*         exit(1); */
/*     } */
}

__device__ u8* device_gpu_ruleset;
u8* gpu_ruleset = NULL;
u8* cpu_ruleset = NULL;

__inline__ __host__ __device__ u8* get_ruleset() {
#ifdef __CUDA_ARCH__
    return device_gpu_ruleset;
#else
    return cpu_ruleset;
#endif
}

// nahrazeno CUDA zdroji
/* __device__ u8* gpu_grid_states_1 = NULL; */
/* __device__ u8* gpu_grid_states_2 = NULL; */

u8 *cpu_grid_states_1 = NULL;
u8 *cpu_grid_states_2 = NULL;
u8 *cpu_grid_states_tmp = NULL;

// udalosti pro mereni casu v CUDA
hipEvent_t start, stop;

__inline__ __host__ __device__ i32 get_cell_index_shared(i32 x, i32 y) {
    assert(x >= 0);
    assert(x < SHARED_SUBGRID_LENGTH);
    assert(y >= 0);
    assert(y < SHARED_SUBGRID_LENGTH);

    return x + y * SHARED_SUBGRID_LENGTH;
}

// returns an index into a 2D row-aligned array
__inline__ __host__ __device__ i32 get_cell_index(i32 x, i32 y) {
    x = mod(x, GRID_WIDTH);
    y = mod(y, GRID_HEIGHT);

    return x + y * GRID_PITCH;
}

__inline__ __host__ __device__ bool cell_state_fit(u8 state_prev, u8 state_next) {
#if FITNESS_FN == FITNESS_FN_STATE_PROPORTION
    return state_next == FITNESS_FN_STATE_PROPORTION_STATE;
#elif FITNESS_FN == FITNESS_FN_UPDATE_PROPORTION
    return state_prev != state_next;
#endif
}

__host__ __device__ u8 get_next_state(u8 current_state, u8* neighbours) {
    u8* ruleset = get_ruleset();

    // In debug mode, validate the `current_state` argument.
    assert(current_state < CELL_STATES);

    // In debug mode, validate the `neighbours` argument.
#ifndef NDEBUG
    {
        u8 total_neighbours = 0;

        for (u8 state = 0; state < CELL_STATES; state++) {
            u8 current_neighbours = neighbours[state];
            total_neighbours += current_neighbours;

            assert(current_neighbours <= CELL_NEIGHBOURHOOD_SIZE);
        }

        assert(total_neighbours == CELL_NEIGHBOURHOOD_SIZE);
    }
#endif

    u32 index = get_rule_index(get_cell_neighbourhood_combinations(), current_state, CELL_STATES, neighbours);

    assert(index < get_ruleset_size());

    return ruleset[index];
}

__host__ __device__ bool update_cell(u8* in_grid, u8* out_grid, i32 x, i32 y) {
    i32 cell_index = get_cell_index(x, y);
    u8 current_state = in_grid[cell_index];
    i32vec2 neighbours[CELL_NEIGHBOURHOOD_SIZE];
    u8 state_count[CELL_STATES] = { 0 };

    get_neighbours(x, y, neighbours);

    for (u32 neighbour_index = 0; neighbour_index < CELL_NEIGHBOURHOOD_SIZE; neighbour_index++) {
        i32vec2 neighbour = neighbours[neighbour_index];
        i32 abs_x = x + neighbour.x;
        i32 abs_y = y + neighbour.y;
        i32 neighbour_cell_index = get_cell_index(abs_x, abs_y);

        state_count[in_grid[neighbour_cell_index]] += 1;
    }

    u8 next_state = get_next_state(current_state, state_count);
    out_grid[cell_index] = next_state;

    return cell_state_fit(current_state, next_state);
}

__device__ bool update_cell_shared(u8* in_subgrid_shared, u8* out_grid, i32 x_global, i32 y_global, i32 x_shared, i32 y_shared) {
    i32 cell_index_shared = get_cell_index_shared(x_shared, y_shared);
    i32 cell_index_global = get_cell_index(x_global, y_global);
    u8 current_state = in_subgrid_shared[cell_index_shared];
    i32vec2 neighbours[CELL_NEIGHBOURHOOD_SIZE];
    u8 state_count[CELL_STATES] = { 0 };

    get_neighbours(x_global, y_global, neighbours);

    for (u32 neighbour_index = 0; neighbour_index < CELL_NEIGHBOURHOOD_SIZE; neighbour_index++) {
        i32vec2 neighbour_offset = neighbours[neighbour_index];
        i32 abs_x_shared = x_shared + neighbour_offset.x;
        i32 abs_y_shared = y_shared + neighbour_offset.y;
        i32 neighbour_cell_index_shared = get_cell_index_shared(abs_x_shared, abs_y_shared);
        u8 neighbour_state = in_subgrid_shared[neighbour_cell_index_shared];
        state_count[neighbour_state] += 1;
    }

    u8 next_state = get_next_state(current_state, state_count);
    out_grid[cell_index_global] = next_state;

    return cell_state_fit(current_state, next_state);
}

/* funkce zajistujici aktualizaci simulace - verze pro CPU
 *  in_grid - vstupni simulacni mrizka
 *  out_grid - vystupni simulacni mrizka
 *  width - sirka simulacni mrizky
 *  height - vyska simulacni mrizky
 */
__host__ void cpu_simulate_step(u8* in_grid, u8* out_grid) {
    for (int y = 0; y < GRID_HEIGHT; y++) {
        for (int x = 0; x < GRID_WIDTH; x++) {
            update_cell(in_grid, out_grid, x, y);
        }
    }
}

__global__ void gpu_simulate_step_kernel_shared(u8* in_grid, u8* out_grid) {
    __shared__ u8 shared_data[SHARED_SUBGRID_AREA];

    // load shared_subgrid
    i32 x0 = blockIdx.x * BLOCK_LENGTH - SHARED_SUBGRID_MARGIN;
    i32 y0 = blockIdx.y * BLOCK_LENGTH - SHARED_SUBGRID_MARGIN;

    for (i32 i = 0; i < SHARED_SUBGRID_LOAD_ITERATIONS; i++) {
        i32 index_shared = threadIdx.x + threadIdx.y * BLOCK_LENGTH + i * BLOCK_AREA;

        if (index_shared < SHARED_SUBGRID_AREA) {
            i32 x = x0 + index_shared % SHARED_SUBGRID_LENGTH;
            i32 y = y0 + index_shared / SHARED_SUBGRID_LENGTH;
            i32 index_global = get_cell_index(x, y);

            shared_data[index_shared] = in_grid[index_global];
        }
    }

    __syncthreads(); // ensure input subgrid is loaded into shared memory before continuing

    i32 x = threadIdx.x + blockIdx.x * BLOCK_LENGTH;
    i32 y = threadIdx.y + blockIdx.y * BLOCK_LENGTH;
    i32 x_shared = threadIdx.x + SHARED_SUBGRID_MARGIN;
    i32 y_shared = threadIdx.y + SHARED_SUBGRID_MARGIN;
    bool write = x < GRID_WIDTH && y < GRID_HEIGHT;
    bool fit = write;

    if (!write) {
        fit = update_cell_shared(shared_data, out_grid, x, y, x_shared, y_shared);
    }

    /*
    Until now, `shared_data` was used for input states.
    Past this `__syncthreads` call, it is used for a parallel reduction of fit states.
     */
    __syncthreads();

    i32 fit_index = threadIdx.x + threadIdx.y * BLOCK_LENGTH;
    shared_fit_cells[fit_index] = (u8) fit;

    // TODO: interpret `shared_data` as `u8*`, `u16*` and `u32*` based on the iteration of the parallel reduction

    __syncthreads();
}

 __global__ void gpu_simulate_step_kernel_noshared(u8* in_grid, u8* out_grid) {
    i32 x = threadIdx.x + blockIdx.x * BLOCK_LENGTH;
    i32 y = threadIdx.y + blockIdx.y * BLOCK_LENGTH;

    if (x < GRID_WIDTH && y < GRID_HEIGHT) {
        update_cell(in_grid, out_grid, x, y);
    }
 }

void simulate_multiple_steps() {

    const i32 STEPS = 1;

    // grid and block dimensions
    dim3 blocks(GRID_WIDTH_IN_BLOCKS, GRID_HEIGHT_IN_BLOCKS);
    dim3 threads(BLOCK_LENGTH, BLOCK_LENGTH);

    CHECK_ERROR(hipGraphicsMapResources(1, &gpu_cuda_grid_states_1, 0));
    CHECK_ERROR(hipGraphicsMapResources(1, &gpu_cuda_grid_states_2, 0));

    u8* gpu_grid_states_1 = NULL;
    u8* gpu_grid_states_2 = NULL;

    CHECK_ERROR(hipGraphicsResourceGetMappedPointer((void**) &gpu_grid_states_1, NULL, gpu_cuda_grid_states_1));
    CHECK_ERROR(hipGraphicsResourceGetMappedPointer((void**) &gpu_grid_states_2, NULL, gpu_cuda_grid_states_2));

    // ulozeni pocatecniho casu
    CHECK_ERROR(hipEventRecord(start, 0));

    // aktualizace simulace + vygenerovani bitmapy pro zobrazeni stavu simulace
    for (i32 i = 0; i < STEPS; i++) {
        if (USE_SHARED_MEMORY) {
            gpu_simulate_step_kernel_shared<<<blocks, threads>>>(gpu_grid_states_1, gpu_grid_states_2);
        } else {
            gpu_simulate_step_kernel_noshared<<<blocks, threads>>>(gpu_grid_states_1, gpu_grid_states_2);
        }

        swap(gpu_vbo_grid_states_1, gpu_vbo_grid_states_2);
        swap(gpu_cuda_grid_states_1, gpu_cuda_grid_states_2);
        swap(gpu_grid_states_1, gpu_grid_states_2);
    }

    // ulozeni casu ukonceni simulace
    CHECK_ERROR(hipEventRecord(stop, 0));
    CHECK_ERROR(hipEventSynchronize(stop));

    float elapsedTime;

    // vypis casu simulace
    CHECK_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Update: %f ms\n", elapsedTime);

#if CPU_VERIFY
    printf("Verifying on the CPU...\n");

    for (i32 i = 0; i < STEPS; i++) {
        // krok simulace life game na CPU
        cpu_simulate_step(cpu_grid_states_1, cpu_grid_states_2);
        swap(cpu_grid_states_1, cpu_grid_states_2);
    }

    hipMemcpy(cpu_grid_states_tmp, gpu_grid_states_1, GRID_AREA_WITH_PITCH * sizeof(u8), hipMemcpyDeviceToHost);

    int diffs = 0;

    // porovnani vysledku CPU simulace a GPU simulace
    for (i32 y = 0; y < GRID_HEIGHT; y++) {
        for (i32 x = 0; x < GRID_WIDTH; x++) {
            i32 cell_index = get_cell_index(x, y);

            if (cpu_grid_states_1[cell_index] != cpu_grid_states_tmp[cell_index]) {
                diffs++;
            }
        }
    }

    if(diffs != 0)
        std::cout << "CHYBA: " << diffs << " rozdily mezi CPU & GPU simulacni mrizkou" << std::endl;
#endif

    CHECK_ERROR(hipGraphicsUnmapResources(1, &gpu_cuda_grid_states_1, 0));
    CHECK_ERROR(hipGraphicsUnmapResources(1, &gpu_cuda_grid_states_2, 0));

}

// called every frame
void idle_func() {
    simulate_multiple_steps();
    glutPostRedisplay();
}

void finalize(void);

static void handle_keys(unsigned char key, int x, int y) {
    switch (key) {
        case 27:	// ESC
            finalize();
            exit(0);
    }
}

// Writes a ruleset to a file
void ruleset_save(u8* ruleset, char* filename) {
    FILE* file = fopen(filename, "wb");

    if (get_ruleset_size() != fwrite(ruleset, sizeof(u8), get_ruleset_size(), file)) {
        fprintf(stderr, "Failed to write a ruleset.");
        exit(1);
    }

    fclose(file);
}

// Loads a ruleset from a file to a pre-allocated buffer
void ruleset_load(u8* ruleset, char* filename) {
    FILE* file = fopen(filename, "rb");

    if (get_ruleset_size() != fread(ruleset, sizeof(u8), get_ruleset_size(), file)) {
        fprintf(stderr, "Failed to load a ruleset.");
        exit(1);
    }

    fclose(file);
}

void ruleset_load_alloc(u8** ruleset, char* filename) {
    *ruleset = (u8*) calloc(get_ruleset_size(), sizeof(u8));
    ruleset_load(*ruleset, filename);
}

// inicializace CUDA - alokace potrebnych dat a vygenerovani pocatecniho stavu lifu
void initialize(int argc, char **argv) {
    init_draw(argc, argv, handle_keys, idle_func);

    // alokovani mista pro bitmapu na GPU
    CHECK_ERROR(hipMalloc((void**) &(gpu_ruleset), get_ruleset_size() * sizeof(u8)));

    cpu_ruleset = (u8*) calloc(get_ruleset_size(), sizeof(u8));
    cpu_grid_states_1 = (u8*) calloc(GRID_AREA_WITH_PITCH, sizeof(u8));
    cpu_grid_states_2 = (u8*) calloc(GRID_AREA_WITH_PITCH, sizeof(u8));
    cpu_grid_states_tmp = (u8*) calloc(GRID_AREA_WITH_PITCH, sizeof(u8));

    srand(0);

    /* for (int i = 0; i < GRID_AREA; i++) { */
    /*     cpu_grid_states_1[i] = (u8) (rand() % CELL_STATES); */
    /* } */
    cpu_grid_states_1[(GRID_HEIGHT / 2) * GRID_PITCH + GRID_WIDTH / 2] = 1;

    CHECK_ERROR(hipGraphicsMapResources(1, &gpu_cuda_grid_states_1, 0));
    CHECK_ERROR(hipGraphicsMapResources(1, &gpu_cuda_grid_states_2, 0));

    u8* gpu_grid_states_1 = NULL;
    u8* gpu_grid_states_2 = NULL;

    CHECK_ERROR(hipGraphicsResourceGetMappedPointer((void**) &gpu_grid_states_1, NULL, gpu_cuda_grid_states_1));
    CHECK_ERROR(hipGraphicsResourceGetMappedPointer((void**) &gpu_grid_states_2, NULL, gpu_cuda_grid_states_2));

    // prekopirovani pocatecniho stavu do GPU
    hipMemcpy(gpu_grid_states_1, cpu_grid_states_1, GRID_AREA_WITH_PITCH * sizeof(u8), hipMemcpyHostToDevice);
    hipMemcpy(gpu_grid_states_2, cpu_grid_states_1, GRID_AREA_WITH_PITCH * sizeof(u8), hipMemcpyHostToDevice);

    CHECK_ERROR(hipGraphicsUnmapResources(1, &gpu_cuda_grid_states_1, 0));
    CHECK_ERROR(hipGraphicsUnmapResources(1, &gpu_cuda_grid_states_2, 0));

    // Initialize ruleset. Keep first rule as 0.
    for (i32 i = 1; i < get_ruleset_size(); i++) {
        cpu_ruleset[i] = (u8) (rand() % CELL_STATES);
    }

    CHECK_ERROR(hipMemcpy(gpu_ruleset, cpu_ruleset, get_ruleset_size() * sizeof(u8), hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(device_gpu_ruleset), &gpu_ruleset, sizeof(u8*)));

    // vytvoreni struktur udalosti pro mereni casu
    CHECK_ERROR(hipEventCreate( &start ));
    CHECK_ERROR(hipEventCreate( &stop ));
}

// funkce volana pri ukonceni aplikace, uvolni vsechy prostredky alokovane v CUDA 
void finalize(void) {
    // uvolneni bitmapy - na CPU i GPU
    hipFree(gpu_ruleset);

    // uvolneni simulacnich mrizek pro CPU variantu lifu
    free(cpu_ruleset);
    free(cpu_grid_states_1);
    free(cpu_grid_states_2);
    free(cpu_grid_states_tmp);

    // zruseni struktur udalosti
    CHECK_ERROR(hipEventDestroy( start ));
    CHECK_ERROR(hipEventDestroy( stop ));

    finalize_draw();
}

int main_simulate(int argc, char **argv) {
    initialize(argc, argv);

    return ui_loop();
}

int main(int argc, char **argv) {
    bool seek = argc >= 2 && strcmp(argv[1], "seek") == 0;
    bool simulate = argc >= 2 && strcmp(argv[1], "simulate") == 0;

    if (!seek && !simulate) {
        printf("Usage:\n");
        printf("%s seek GRID.rsg -- performs search for interesting rulesets\n", argv[0]);
        printf("%s simulate GRID.rsg RULESET.rsr -- performs visual simulation of an existing ruleset\n", argv[0]);
        exit(0);
    }

    print_configuration();

    if (PROMPT_TO_START) {
        printf("Press Enter to begin.");
        getchar();
    }

    if (simulate) {
        return main_simulate(argc, argv);
    }

    return 0;
}
