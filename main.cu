#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <GL/glut.h>
#include "util.cuh"
#include "draw.cuh"
#include "math.cuh"

using namespace std;

// All possible grid geometries
#define GRID_GEOMETRY_SQUARE   0 // square tiling, like conway's game of life
#define GRID_GEOMETRY_TRIANGLE 1 // triangular tiling
#define GRID_GEOMETRY_HEXAGON  2 // hexagonal tiling

// All possible cell neighbourhoods
#define CELL_NEIGHBOURHOOD_TYPE_VERTEX 0 // a cell is in the current cell's neighbourhood iff it shares a vertex
#define CELL_NEIGHBOURHOOD_TYPE_EDGE   1 // a cell is in the current cell's neighbourhood iff it shares an edge


/**************************
 * START OF CONFIGURATION *
 **************************/

// [uint] simulation grid width
#define GRID_WIDTH  768
// [uint] simulation grid height
#define GRID_HEIGHT 768
// [enum] the shape of the grid's cells (square for Conway's GoL)
/* #define GRID_GEOMETRY GRID_GEOMETRY_TRIANGLE */
// [enum] which cells are considered in the neighbourhood (vertex for Conway's GoL)
/* #define CELL_NEIGHBOURHOOD_TYPE CELL_NEIGHBOURHOOD_TYPE_VERTEX */
// [uchar] number of states a cell can become (2 for Conway's GoL)
/* #define CELL_STATES 8 */

// Conway's GoL:
#define GRID_GEOMETRY GRID_GEOMETRY_SQUARE
#define CELL_NEIGHBOURHOOD_TYPE CELL_NEIGHBOURHOOD_TYPE_VERTEX
#define CELL_STATES 2

// [uint] execution block width and height
#define BLOCK_LENGTH 16
// [bool] whether to verify the GPU simulation with an equivalent CPU simulation
#define CPU_VERIFY IS_DEBUG

/************************
 * END OF CONFIGURATION *
 ************************/


// Automatically derived constants from the configuration
#if GRID_GEOMETRY == GRID_GEOMETRY_SQUARE
    #if CELL_NEIGHBOURHOOD_TYPE == CELL_NEIGHBOURHOOD_TYPE_VERTEX
        #define CELL_NEIGHBOURHOOD_SIZE 8
    #elif CELL_NEIGHBOURHOOD_TYPE == CELL_NEIGHBOURHOOD_TYPE_EDGE
        #define CELL_NEIGHBOURHOOD_SIZE 4
    #endif
#elif GRID_GEOMETRY == GRID_GEOMETRY_TRIANGLE
    #if CELL_NEIGHBOURHOOD_TYPE == CELL_NEIGHBOURHOOD_TYPE_VERTEX
        #define CELL_NEIGHBOURHOOD_SIZE 12
    #elif CELL_NEIGHBOURHOOD_TYPE == CELL_NEIGHBOURHOOD_TYPE_EDGE
        #define CELL_NEIGHBOURHOOD_SIZE 4
    #endif
#elif GRID_GEOMETRY == GRID_GEOMETRY_HEXAGON
    #if CELL_NEIGHBOURHOOD_TYPE == CELL_NEIGHBOURHOOD_TYPE_VERTEX
        #define CELL_NEIGHBOURHOOD_SIZE 6
    #elif CELL_NEIGHBOURHOOD_TYPE == CELL_NEIGHBOURHOOD_TYPE_EDGE
        #define CELL_NEIGHBOURHOOD_SIZE 6
    #endif
#endif

#define BLOCK_AREA            (BLOCK_LENGTH * BLOCK_LENGTH)                     // threads per execution block
#define GRID_WIDTH_IN_BLOCKS  ((GRID_WIDTH + BLOCK_LENGTH - 1) / BLOCK_LENGTH)  // execution grid width
#define GRID_HEIGHT_IN_BLOCKS ((GRID_HEIGHT + BLOCK_LENGTH - 1) / BLOCK_LENGTH) // execution grid height

// Capitalised because they are effectively constant
int CPU_CELL_NEIGHBOURHOOD_COMBINATIONS = -1;
__constant__ int GPU_CELL_NEIGHBOURHOOD_COMBINATIONS = -1;
int CPU_RULESET_SIZE = -1;
__constant__ int GPU_RULESET_SIZE = -1;

__inline__ __host__ __device__ int get_cell_neighbourhood_combinations() {
#ifdef __CUDA_ARCH__
    return GPU_CELL_NEIGHBOURHOOD_COMBINATIONS;
#else
    return CPU_CELL_NEIGHBOURHOOD_COMBINATIONS;
#endif
}

__inline__ __host__ __device__ int get_ruleset_size() {
#ifdef __CUDA_ARCH__
    return GPU_RULESET_SIZE;
#else
    return CPU_RULESET_SIZE;
#endif
}

void print_configuration() {
    printf("\nConfiguration:\n");
    printf("\tGrid width: %d\n", GRID_WIDTH);
    printf("\tGrid height: %d\n", GRID_HEIGHT);

    if (GRID_GEOMETRY == GRID_GEOMETRY_SQUARE) {
        printf("\tGrid geometry: Square\n");
    } else if (GRID_GEOMETRY == GRID_GEOMETRY_TRIANGLE) {
        printf("\tGrid geometry: Triangle\n");
    } else if (GRID_GEOMETRY == GRID_GEOMETRY_HEXAGON) {
        printf("\tGrid geometry: Hexagon\n");
    } else {
        printf("\tGrid geometry: Invalid, aborting...\n");
        exit(1);
    }

    if (CELL_NEIGHBOURHOOD_TYPE == CELL_NEIGHBOURHOOD_TYPE_VERTEX) {
        printf("\tCell neighbourhood type: Vertex\n");
    } else if (CELL_NEIGHBOURHOOD_TYPE == CELL_NEIGHBOURHOOD_TYPE_EDGE) {
        printf("\tCell neighbourhood type: Edge\n");
    } else {
        printf("\tCell neighbourhood type: Invalid, aborting...\n");
        exit(1);
    }

    printf("\tCell neighbourhood size: %d\n", CELL_NEIGHBOURHOOD_SIZE);
    printf("\tCell states: %d\n", CELL_STATES);

    CPU_CELL_NEIGHBOURHOOD_COMBINATIONS = compute_neighbouring_state_combinations(CELL_NEIGHBOURHOOD_SIZE, CELL_STATES);
    hipMemcpyToSymbol(HIP_SYMBOL(GPU_CELL_NEIGHBOURHOOD_COMBINATIONS), &CPU_CELL_NEIGHBOURHOOD_COMBINATIONS, sizeof(int));
    printf("\tCell neighbourhood combinations: %d (with a combinatorial number system, %ld with simple indexing)\n", CPU_CELL_NEIGHBOURHOOD_COMBINATIONS, powli(CELL_NEIGHBOURHOOD_SIZE, CELL_STATES));

    CPU_RULESET_SIZE = compute_ruleset_size(CELL_NEIGHBOURHOOD_SIZE, CELL_STATES);
    hipMemcpyToSymbol(HIP_SYMBOL(GPU_RULESET_SIZE), &CPU_RULESET_SIZE, sizeof(int));
    printf("\tRuleset size: %d (with a combinatorial number system, %ld with simple indexing)\n", CPU_RULESET_SIZE, CELL_STATES * powli(CELL_NEIGHBOURHOOD_SIZE, CELL_STATES));
    printf("\n");
}

__device__ u8* device_gpu_ruleset;
u8* gpu_ruleset = NULL;
u8* cpu_ruleset = NULL;

__device__ u8* gpu_grid_states_1 = NULL;
__device__ u8* gpu_grid_states_2 = NULL;

u8 *cpu_grid_states_1 = NULL;
u8 *cpu_grid_states_2 = NULL;
u8 *cpu_grid_states_tmp = NULL;

// udalosti pro mereni casu v CUDA
hipEvent_t start, stop;

__inline__ __host__ __device__ int getCellIndex(int width, int height, int x, int y) {
    x = mod(x, width);
    y = mod(y, height);

    return x + y * width;
}

__host__ __device__ u8 getNextState(u8 current_state, u8* neighbours) {
#ifdef __CUDA_ARCH__
    u8* ruleset = device_gpu_ruleset;
#else
    u8* ruleset = cpu_ruleset;
#endif

    // In debug mode, validate the `current_state` argument.
    assert(current_state < CELL_STATES);

    // In debug mode, validate the `neighbours` argument.
#ifndef NDEBUG
    {
        u8 total_neighbours = 0;

        for (u8 state = 0; state < CELL_STATES; state++) {
            u8 current_neighbours = neighbours[state];
            total_neighbours += current_neighbours;

            if (!(current_neighbours <= CELL_NEIGHBOURHOOD_SIZE)) {
                printf("current_neighbours: %d\n", current_neighbours);
                assert(current_neighbours <= CELL_NEIGHBOURHOOD_SIZE);
            }
        }

        assert(total_neighbours == CELL_NEIGHBOURHOOD_SIZE);
    }
#endif

    /* int index = current_state * 9 + neighbours; */
    int index = get_rule_index(get_cell_neighbourhood_combinations(), current_state, CELL_STATES, neighbours);

    if (index >= get_ruleset_size()) {
        printf("index: %d\n", index);
        printf("gpu_ruleset_size: %d\n", get_ruleset_size());
    }

    assert(index >= 0);
    assert(index < get_ruleset_size());

    return ruleset[index];
}

__host__ __device__ u8 updateCell(u8* in, u8* out, int width, int height, int x, int y) {
    int cellID = getCellIndex(width, height, x, y);
    u8 currentState = in[cellID];
    u8 neighbours[CELL_STATES] = { 0 };

    for (int rel_y = -1; rel_y <= 1; rel_y++) {
        for (int rel_x = -1; rel_x <= 1; rel_x++) {
            if (rel_x == 0 && rel_y == 0) {
                continue;
            }

            int abs_x = x + rel_x;
            int abs_y = y + rel_y;
            int neighbourID = getCellIndex(width, height, abs_x, abs_y);

            neighbours[in[neighbourID]] += 1;
        }
    }

    int nextState = getNextState(currentState, neighbours);
    out[cellID] = nextState;

    return nextState;
}

/* funkce zajistujici aktualizaci simulace - verze pro CPU
 *  in - vstupni simulacni mrizka
 *  out - vystupni simulacni mrizka
 *  width - sirka simulacni mrizky
 *  height - vyska simulacni mrizky
 */
void life_cpu(u8* in, u8* out, int width, int height) {
    int threadID;

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            updateCell(in, out, width, height, x, y);
        }
    }
}

// funkce pro zapis barvy pixelu do bitmapy, nova barva je odvozena ze stavu simulace
__inline__ __device__ void stateToColor(u8 oldValue, u8 newValue, uchar4* bitmap, int bitmapId) {
    uchar4 color;

    color.x = (newValue==0 && oldValue==1) ? 255 : 0;
    color.y = (newValue==1 && oldValue==0) ? 255 : 0;
    color.z = (newValue==1 && oldValue==1) ? 255 : 0;
    color.w = 0;

    bitmap[bitmapId] = color;
}

__global__ void life_kernel(uchar4* bitmap, u8* in, u8* out, int width, int height) {
    int x = threadIdx.x + blockIdx.x * BLOCK_LENGTH;
    int y = threadIdx.y + blockIdx.y * BLOCK_LENGTH;
    int threadID = getCellIndex(width, height, x, y);

    if (threadID < width*height) {
        u8 oldValue = in[threadID];
        u8 newValue = updateCell(in, out, width, height, x, y);

        stateToColor(oldValue, newValue, bitmap, threadID);
    }

    /* int combination[3] = { */
    /*     5, 0, 3 */
    /* }; */

    /* if (threadID == 0) { */
    /*     int result = combination_index_with_repetition(3, combination); */
    /*     printf("gpu result: %d\n", result); */
    /* } */
}


// funkce pro spusteni kernelu + priprava potrebnych dat a struktur
void callKernelCUDA(void) {
    // ulozeni pocatecniho casu
    CHECK_ERROR(hipEventRecord(start, 0));

    // grid and block dimensions
    dim3 blocks(GRID_WIDTH_IN_BLOCKS, GRID_HEIGHT_IN_BLOCKS);
    dim3 threads(BLOCK_LENGTH, BLOCK_LENGTH);

    // aktualizace simulace + vygenerovani bitmapy pro zobrazeni stavu simulace
    life_kernel<<<blocks,threads>>>(bitmap->deviceData, gpu_grid_states_1, gpu_grid_states_2, bitmap->width, bitmap->height);

    // prohozeni ukazatelu (u textur pouzit pouze gpu_grid_states_2)
    swap(gpu_grid_states_1, gpu_grid_states_2);

    // ulozeni casu ukonceni simulace
    CHECK_ERROR(hipEventRecord(stop, 0));
    CHECK_ERROR(hipEventSynchronize(stop));

    float elapsedTime;

    // vypis casu simulace
    CHECK_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Update: %f ms\n", elapsedTime);

    // kopirovani bitmapy zpet na CPU pro zobrazeni
    CHECK_ERROR(hipMemcpy(bitmap->pixels, bitmap->deviceData, bitmap->width*bitmap->height*sizeof(uchar4), hipMemcpyDeviceToHost));

#if CPU_VERIFY
    printf("Verifying on the CPU...\n");
    // krok simulace life game na CPU
    life_cpu(cpu_grid_states_1, cpu_grid_states_2, bitmap->width, bitmap->height);
    swap(cpu_grid_states_1, cpu_grid_states_2);

    hipMemcpy(cpu_grid_states_tmp, gpu_grid_states_1, bitmap->width*bitmap->height*sizeof(u8), hipMemcpyDeviceToHost);

    int diffs = 0;

    // porovnani vysledku CPU simulace a GPU simulace
    for(int row=0;row<bitmap->height;row++) {
        for(int col=0; col<bitmap->width; col++) {

            int rowAddr = row * bitmap->width;
            int threadID = rowAddr + col;	// index vlakna

            if(cpu_grid_states_1[threadID] != cpu_grid_states_tmp[threadID])
                diffs++;
        }
    }

    if(diffs != 0)
        std::cout << "CHYBA: " << diffs << " rozdily mezi CPU & GPU simulacni mrizkou" << std::endl;
#endif
}


// inicializace CUDA - alokace potrebnych dat a vygenerovani pocatecniho stavu lifu
void initialize(void) {
    // alokace struktury bitmapy
    bitmap = (bitmap_t*) malloc(sizeof(bitmap));
    bitmap->width = GRID_WIDTH;
    bitmap->height = GRID_HEIGHT;

    hipHostAlloc((void**) &(bitmap->pixels), bitmap->width*bitmap->height*sizeof(uchar4), hipHostMallocDefault);


    // alokovani mista pro bitmapu na GPU
    int bitmapSize = bitmap->width*bitmap->height;
    CHECK_ERROR(hipMalloc((void**) &(gpu_ruleset), 2 * 9 * sizeof(u8)));
    CHECK_ERROR(hipMalloc((void**) &(bitmap->deviceData), bitmapSize*sizeof(uchar4)));
    CHECK_ERROR(hipMalloc((void**) &(gpu_grid_states_1), bitmapSize*sizeof(u8)));
    CHECK_ERROR(hipMalloc((void**) &(gpu_grid_states_2), bitmapSize*sizeof(u8)));

    hipMemset(bitmap->deviceData, 0, bitmapSize*sizeof(uchar4));

    cpu_ruleset = (u8*) malloc(2 * 9 * sizeof(u8));
    cpu_grid_states_1 = (u8*) malloc(bitmapSize*sizeof(u8));
    cpu_grid_states_2 = (u8*) malloc(bitmapSize*sizeof(u8));
    cpu_grid_states_tmp = (u8*) malloc(bitmapSize*sizeof(u8));

    srand(0);

    // inicializace pocatecniho stavu lifu
    for (int i = 0; i < bitmapSize; i++) {
        cpu_grid_states_1[i] = (u8) (rand() % 2);
    }

    // prekopirovani pocatecniho stavu do GPU
    hipMemcpy(gpu_grid_states_1, cpu_grid_states_1, bitmapSize*sizeof(u8), hipMemcpyHostToDevice);

    // nakopirovani tabulky novych stavu do konstantni pameti
    u8 ruleset[2 * 9] = {
        // currentState == 0:
        0, 0, 0, 1, 0, 0, 0, 0, 0,
        // currentState == 1:
        0, 0, 1, 1, 0, 0, 0, 0, 0,
    };

    memcpy(cpu_ruleset, ruleset, 2 * 9 * sizeof(u8));
    CHECK_ERROR(hipMemcpy(gpu_ruleset, ruleset, 2 * 9 * sizeof(u8), hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(device_gpu_ruleset), &gpu_ruleset, sizeof(u8*)));

    // vytvoreni struktur udalosti pro mereni casu
    CHECK_ERROR(hipEventCreate( &start ));
    CHECK_ERROR(hipEventCreate( &stop ));
}

// funkce volana pri ukonceni aplikace, uvolni vsechy prostredky alokovane v CUDA 
void finalize(void) {

    // uvolneni bitmapy - na CPU i GPU
    if (bitmap != NULL) {
        if (bitmap->pixels != NULL) {
            // uvolneni bitmapy na CPU
            hipHostFree(bitmap->pixels);
            bitmap->pixels = NULL;
        }
        if (bitmap->deviceData != NULL) {
            // uvolneni bitmapy na GPU
            hipFree(bitmap->deviceData);
            bitmap->deviceData = NULL;
        }
        hipFree(gpu_ruleset);
        hipFree(gpu_grid_states_1);
        hipFree(gpu_grid_states_2);
        free(bitmap);
    }

    // uvolneni simulacnich mrizek pro CPU variantu lifu
    free(cpu_ruleset);
    free(cpu_grid_states_1);
    free(cpu_grid_states_2);
    free(cpu_grid_states_tmp);

    // zruseni struktur udalosti
    CHECK_ERROR(hipEventDestroy( start ));
    CHECK_ERROR(hipEventDestroy( stop ));
}

// called every frame
void idle_func() {
    callKernelCUDA();
    glutPostRedisplay();
}

static void handle_keys(unsigned char key, int x, int y) {
    switch (key) {
        case 27:	// ESC
            finalize();
            exit(0);
    }
}

int main(int argc, char **argv) {
    print_configuration();
    initialize();

    printf("Press Enter to begin simulation.");
    getchar();

    return ui_loop(argc, argv, GRID_WIDTH, GRID_HEIGHT, handle_keys, idle_func);
}
